#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h> // import to make my LSP happy
/* Problem size */
#define XSIZE 2560
#define YSIZE 2048
/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32
#define MAXITER 255 // you may want to increase this
double xleft=-2.01;
double xright=1;
double yupper,ylower;
double ycenter=1e-6;
double step;
int host_pixel[XSIZE*YSIZE];
int device_pixel[XSIZE*YSIZE];
typedef struct {
    float real,imag; // needed to use float not double
} my_complex_t;

#define PIXEL(i,j) ((i)+(j)*XSIZE)

// ********** SUBTASK1: Create kernel device_calculate ******************/
__global__ 
void device_calculate(double xleft, double yupper, double step, int* output) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= YSIZE || i >= XSIZE) return; // because of ceil in grid dimension, might overshoot
    my_complex_t c,z,temp;
    int iter=0;
    c.real = (xleft + step*i);
    c.imag = (yupper - step*j);
    z = c;
    while(z.real*z.real + z.imag*z.imag<4.0) {
        temp.real = z.real*z.real - z.imag*z.imag + c.real;
        temp.imag = 2.0*z.real*z.imag + c.imag;
        z = temp;
        if(++iter==MAXITER) break;
    }
    output[PIXEL(i, j)]=iter;
}
// BLOCKX * BLOCKY threads per block
// total number of blocks are calculated using ceiling, to make sure we cover everything
// ceil(a / b) == floor((a + b - 1) / b) == (a + b - 1) / b (when truncating)
dim3 gridDimensions = {(XSIZE + BLOCKX - 1) / BLOCKX, (YSIZE + BLOCKY - 1) / BLOCKY, 1}; 
dim3 blockDimensions = {BLOCKX, BLOCKY, 1};
// ********** SUBTASK1 END ***********************************************/

void host_calculate() {
    for(int j=0;j<YSIZE;j++) {
        for(int i=0;i<XSIZE;i++) {
            /* Calculate the number of iterations until divergence for
               each pixel.
               If divergence never happens, return MAXITER */
            my_complex_t c,z,temp;
            int iter=0;
            c.real = (xleft + step*i);
            c.imag = (yupper - step*j);
            z = c;
            while(z.real*z.real + z.imag*z.imag<4.0) {
                temp.real = z.real*z.real - z.imag*z.imag + c.real;
                temp.imag = 2.0*z.real*z.imag + c.imag;
                z = temp;
                if(++iter==MAXITER) break;
            }
            host_pixel[PIXEL(i,j)]=iter;
        }
    }
}

typedef unsigned char uchar;

// save 24-bits bmp file, buffer must be in bmp format: upside-down
void savebmp(const char *name,uchar *buffer,int x,int y) {
    FILE *f=fopen(name,"wb");
    if(!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size=x*y*3+54;
    uchar
        header[54]={'B','M',uchar(size&255),uchar((size>>8)&255),uchar((size>>16)&255),uchar(size>>24),0,
            0,0,0,54,0,0,0,40,0,0,0,uchar(x&255),uchar(x>>8),0,0,uchar(y&255),uchar(y>>8),0,0,1,0,24,0,0,0,0,0,0
                ,
            0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    fwrite(header,1,54,f);
    fwrite(buffer,1,x*y*3,f);
    fclose(f);
}

// given iteration number, set a color
void fancycolour(uchar *p,int iter) {
    if(iter==MAXITER);
    else if(iter<8) { p[0]=128+iter*16; p[1]=p[2]=0; }
    else if(iter<24) { p[0]=255; p[1]=p[2]=(iter-8)*16; }
    else if(iter<160) { p[0]=p[1]=255-(iter-24)*2; p[2]=255; }
    else { p[0]=p[1]=(iter-160)*2; p[2]=255-(iter-160)*2; }
}
// Get system time to microsecond precision
// ostensibly, similar to MPI_Wtime),
// returns time in seconds
double walltime ( void ) {
    static struct timeval t;
    gettimeofday ( &t, NULL );
    return ( t.tv_sec + 1e-6 * t.tv_usec );
}
int main(int argc,char **argv) {
    if(argc==1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1=yes, 0=no)");
        return 0;
    }
    double start;
    double hosttime=0;
    double devicetime=0;
    double memtime=0;
    hipDeviceProp_t p;
    hipSetDevice(0);
    hipGetDeviceProperties (&p, 0);
    printf("Device compute capability: %d.%d\n", p.major, p.minor);
    /* Calculate the range in the y-axis such that we preserve the
       aspect ratio */
    step=(xright-xleft)/XSIZE;
    yupper=ycenter+(step*YSIZE)/2;
    ylower=ycenter-(step*YSIZE)/2;
    /* Host calculates image */
    start=walltime();
    host_calculate();
    hosttime+=walltime()-start;
    //********** SUBTASK2: Set up device memory ***************************/

    int* device_output;
    hipError_t err = hipMallocManaged(&device_output, XSIZE * YSIZE * sizeof(int));
    if (err != hipSuccess) {
        printf("Error in cudaMalloc\n");
    }
    /********** SUBTASK2 END **********************************************/

    start=walltime();
    //********* SUBTASK3: Execute the kernel on the device ************/
    device_calculate<<<gridDimensions, blockDimensions>>>(xleft, yupper, step, device_output);
    //********** SUBTASK3 END *****************************************/
    devicetime+=walltime()-start;
    start=walltime();
    //***** SUBTASK4: Transfer the result from device to device_pixel[][]*/
    err = hipMemcpy(device_pixel, device_output, XSIZE * YSIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error in cudaMemcpy\n");
    }
    //********** SUBTASK4 END ******************************************/
    memtime+=walltime()-start;
    /****** SUBTASK5: Free the device memory also ************************/
    hipFree(device_output);
    /********** SUBTASK5 END ******************************************/
    int errors=0;
    /* check if result is correct */
    for(int i=0;i<XSIZE;i++) {
        for(int j=0;j<YSIZE;j++) {
            int diff=host_pixel[PIXEL(i,j)]-device_pixel[PIXEL(i,j)];
            if(diff<0) diff=-diff;
            /* allow +-1 difference */
            if(diff>1) {
                if(errors<10) printf("Error on pixel %d %d: expected %d, found %d\n", i,j,host_pixel[PIXEL(i,j)],device_pixel[PIXEL(i,j)]);
                else if(errors==10) puts("...");
                errors++;
            }
        }
    }
    if(errors > 0) printf("Found %d errors.\n",errors);
    else puts("Device calculations are correct.");
    printf("\n");
    printf("Host time: %7.3f ms\n",hosttime*1e3);
    printf("Device calculation: %7.3f ms\n",devicetime*1e3);
    printf("Copy result: %7.3f ms\n",memtime*1e3);
    if(strtol(argv[1],NULL,10)!=0) {
        /* create nice image from iteration counts. take care to create it
           upside
           down (bmp format) */
        { // store device image
            unsigned char *buffer=(unsigned char *)calloc(XSIZE*YSIZE*3,1);
            for(int i=0;i<XSIZE;i++) {
                for(int j=0;j<YSIZE;j++) {
                    int p=((YSIZE-j-1)*XSIZE+i)*3;
                    fancycolour(buffer+p,device_pixel[PIXEL(i,j)]);
                }
            }
            /* write image to disk */
            savebmp("mandel1_device.bmp",buffer,XSIZE,YSIZE);
        }
        { // store host image
            unsigned char *buffer=(unsigned char *)calloc(XSIZE*YSIZE*3,1);
            for(int i=0;i<XSIZE;i++) {
                for(int j=0;j<YSIZE;j++) {
                    int p=((YSIZE-j-1)*XSIZE+i)*3;
                    fancycolour(buffer+p,host_pixel[PIXEL(i,j)]);
                }
            }
            /* write image to disk */
            savebmp("mandel1_host.bmp",buffer,XSIZE,YSIZE);
        }
    }
    return 0;
}
